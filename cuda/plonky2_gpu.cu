#include "hip/hip_runtime.h"
#include "plonky2_gpu_impl.cuh"

struct CudaInvContext {
    hipStream_t stream;
    hipStream_t stream2;
};
template <class T>
struct DataSlice{
    T* ptr;
    int len;
};


#ifndef __CUDA_ARCH__
#include <string>
#include <mutex>

struct RustError { /* to be returned exclusively by value */
    int code;
    char *message;
#ifdef __cplusplus
    RustError(int e = 0) : code(e)
    {   message = nullptr;   }
    RustError(int e, const std::string& str) : code(e)
    {   message = str.empty() ? nullptr : strdup(str.c_str());   }
    RustError(int e, const char *str) : code(e)
    {   message = str==nullptr ? nullptr : strdup(str);   }
    // no destructor[!], Rust takes care of the |message|
#endif
};

#define CUDA_ASSERT(expr) \
do {\
    if (auto code = expr; code != hipSuccess) {\
        printf("%s@%d failed: %s\n", #expr, __LINE__, hipGetErrorString(code));\
        return RustError{code};\
    }\
} while(0)

#include <fstream>
#include <vector>

extern "C" {
    RustError ifft(
            GoldilocksField* d_values_flatten,
            int poly_num, int values_num_per_poly, int log_len,
            const GoldilocksField* d_root_table,
            GoldilocksField* p_inv,
            CudaInvContext* ctx
    ) {
        GoldilocksField n_inv = *p_inv;
        auto stream = ctx->stream;

        clock_t start = clock();
        ifft_kernel<<<poly_num, 32*8, 0, stream>>>(d_values_flatten, poly_num, values_num_per_poly, log_len, d_root_table, n_inv);
        hipStreamSynchronize(stream);
        /* printf("ifft_kernel elapsed: %.2lf\n", (double )(clock()-start) / CLOCKS_PER_SEC * 1000); */

        return RustError{hipSuccess};
    }

    RustError fft_blinding(
            GoldilocksField* d_values_flatten,
            GoldilocksField* d_ext_values_flatten,
            int poly_num, int values_num_per_poly, int log_len,
            const GoldilocksField* d_root_table2, const GoldilocksField* d_shift_powers,
            int rate_bits,
            int pad_extvalues_len,
            CudaInvContext* ctx
    ) {
        auto stream = ctx->stream;

        int thcnt = 0;
        int nthreads = 32;

        d_ext_values_flatten += pad_extvalues_len;


        clock_t start = clock();
        thcnt = values_num_per_poly * poly_num;
        nthreads = 32;
        lde_kernel<<<(thcnt + nthreads - 1) / nthreads, nthreads, 0, stream>>>(d_values_flatten, d_ext_values_flatten,
                                                                               poly_num, values_num_per_poly, rate_bits);
        hipStreamSynchronize(stream);
        printf("lde_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly * poly_num;
        nthreads = 32;
        init_lde_kernel<<<(thcnt + nthreads - 1) / nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num,
                                                                                    values_num_per_poly, rate_bits);
        hipStreamSynchronize(stream);
        printf("init_lde_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly * poly_num;
        nthreads = 32;
        mul_shift_kernel<<<(thcnt + nthreads - 1) / nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num,
                                                                                     values_num_per_poly, rate_bits,
                                                                                     d_shift_powers);
        hipStreamSynchronize(stream);
        printf("mul_shift_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        fft_kernel<<<poly_num, 32 * 8, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly * (1 << rate_bits),
                                                    log_len + rate_bits, d_root_table2, rate_bits);
        hipStreamSynchronize(stream);
        printf("fft_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);
    }


    RustError build_merkle_tree(
        GoldilocksField* d_ext_values_flatten,
        int poly_num, int values_num_per_poly, int log_len,
        int rate_bits, int salt_size,
        int cap_height,
        int pad_extvalues_len,
        CudaInvContext* ctx
    ) {
        int values_num_per_extpoly = values_num_per_poly * (1 << rate_bits);
        auto stream = ctx->stream;
        int ext_poly_num = poly_num + salt_size;

        int len_cap = 1 << cap_height;
        int num_digests = 2 * (values_num_per_extpoly - len_cap);

        int thcnt = 0;
        int nthreads = 32;
        d_ext_values_flatten += pad_extvalues_len;


        clock_t start = clock();
        thcnt = values_num_per_extpoly * poly_num;
        nthreads = 32;
        reverse_index_bits_kernel<<<(thcnt + nthreads - 1) / nthreads, nthreads, 0, stream>>>(d_ext_values_flatten,
                                                                                              poly_num,
                                                                                              values_num_per_extpoly,
                                                                                              log_len + rate_bits);
        hipStreamSynchronize(stream);
        printf("reverse_index_bits_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);

        int log2_leaves_len = log_len + rate_bits;
        assert(cap_height <= log2_leaves_len);

        auto *d_digest_buf = (PoseidonHasher::HashOut *) (d_ext_values_flatten + values_num_per_extpoly * ext_poly_num);

        start = clock();
        thcnt = values_num_per_extpoly;
        nthreads = 32;
        hash_leaves_kernel<<<(thcnt + nthreads - 1) / nthreads, nthreads, 0, stream>>>(
                d_ext_values_flatten, poly_num + salt_size, values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
        hipStreamSynchronize(stream);
        printf("hash_leaves_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        nthreads = 32 * 8;
        thcnt = len_cap * nthreads;
        reduce_digests_kernel<<<(thcnt + nthreads - 1) / nthreads, nthreads, 0, stream>>>(values_num_per_extpoly,
                                                                                          d_digest_buf, len_cap,
                                                                                          num_digests);
        hipStreamSynchronize(stream);
        printf("reduce_digests_kernel elapsed: %.2lf\n", (double) (clock() - start) / CLOCKS_PER_SEC * 1000);
    }


    RustError transpose(
            GoldilocksField* d_ext_values_flatten,
            int poly_num, int values_num_per_poly,
            int rate_bits, int salt_size,
            int pad_extvalues_len,
            CudaInvContext* ctx
    ){
        int values_num_per_extpoly = values_num_per_poly*(1<<rate_bits);
        auto stream = ctx->stream;
        int ext_poly_num = poly_num + salt_size;

        int thcnt = 0;
        int nthreads = 32;
        d_ext_values_flatten += pad_extvalues_len;


        clock_t start = clock();
        thcnt = values_num_per_extpoly;
        nthreads = 32;
        transpose_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, d_ext_values_flatten - pad_extvalues_len, ext_poly_num, values_num_per_extpoly);
        hipStreamSynchronize(stream);
        printf("transpose_kernel elapsed: %.2lf\n",  (double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        return RustError{hipSuccess};
    }

    RustError merkle_tree_from_values(
            GoldilocksField* d_values_flatten,
            GoldilocksField* d_ext_values_flatten,
            int poly_num, int values_num_per_poly, int log_len,
           const GoldilocksField* d_root_table, const GoldilocksField* d_root_table2, const GoldilocksField* d_shift_powers,
           GoldilocksField* p_inv, int rate_bits, int salt_size,
           int cap_height,
           int pad_extvalues_len,
           CudaInvContext* ctx
    ){
        assert(0);
        printf("start merkle_tree_from_values: poly_num:%d, values_num_per_poly:%d, log_len:%d, n_inv:%lu\n",
               poly_num, values_num_per_poly, log_len, p_inv->data);
        printf("d_values_flatten: %p, d_ext_values_flatten: %p\n", d_values_flatten, d_ext_values_flatten);

        int values_num_per_extpoly = values_num_per_poly*(1<<rate_bits);
        GoldilocksField n_inv = *p_inv;
        auto stream = ctx->stream;
        int ext_poly_num = poly_num + salt_size;

        int len_cap = 1 << cap_height;
        int num_digests = 2 * (values_num_per_extpoly - len_cap);

        int thcnt = 0;
        int nthreads = 32;
        double ifft_kernel_use, lde_kernel_use, mul_shift_kernel_use, fft_kernel_use, reverse_index_bits_kernel_use,
                hash_leaves_kernel_use, reduce_digests_kernel_use, transpose_kernel_use;

        d_ext_values_flatten += pad_extvalues_len;
        clock_t start = clock();
        ifft_kernel<<<poly_num, 32*8, 0, stream>>>(d_values_flatten, poly_num, values_num_per_poly, log_len, d_root_table, n_inv);
        hipStreamSynchronize(stream);
        printf("ifft_kernel elapsed: %.2lf\n", ifft_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly*poly_num;
        nthreads = 32;
        lde_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_values_flatten, d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits);
        hipStreamSynchronize(stream);
        printf("lde_kernel elapsed: %.2lf\n", lde_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly*poly_num;
        nthreads = 32;
        init_lde_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits);
        hipStreamSynchronize(stream);
        printf("init_lde_kernel elapsed: %.2lf\n", (double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly*poly_num;
        nthreads = 32;
        mul_shift_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits, d_shift_powers);
        hipStreamSynchronize(stream);
        printf("mul_shift_kernel elapsed: %.2lf\n", mul_shift_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        fft_kernel<<<poly_num, 32*8, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly*(1<<rate_bits), log_len+rate_bits, d_root_table2, rate_bits);
        hipStreamSynchronize(stream);
        printf("fft_kernel elapsed: %.2lf\n", fft_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_extpoly*poly_num;
        nthreads = 32;
        reverse_index_bits_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_extpoly, log_len+rate_bits);
        hipStreamSynchronize(stream);
        printf("reverse_index_bits_kernel elapsed: %.2lf\n", reverse_index_bits_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        int log2_leaves_len = log_len + rate_bits;
        assert(cap_height <= log2_leaves_len);

        auto *d_digest_buf = (PoseidonHasher::HashOut*)(d_ext_values_flatten + values_num_per_extpoly * ext_poly_num);

        start = clock();
        thcnt = values_num_per_extpoly;
        nthreads = 32;
        hash_leaves_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(
                d_ext_values_flatten, poly_num+salt_size, values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
        hipStreamSynchronize(stream);
        printf("hash_leaves_kernel elapsed: %.2lf\n", hash_leaves_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        nthreads = 32*8;
        thcnt = len_cap * nthreads;
        reduce_digests_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
        hipStreamSynchronize(stream);
        printf("reduce_digests_kernel elapsed: %.2lf\n", reduce_digests_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_extpoly;
        nthreads = 32;
        transpose_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, d_ext_values_flatten - pad_extvalues_len, ext_poly_num, values_num_per_extpoly);
        hipStreamSynchronize(stream);
        printf("transpose_kernel elapsed: %.2lf\n", transpose_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        double total_use =
                ifft_kernel_use+
                lde_kernel_use+
                mul_shift_kernel_use+
                fft_kernel_use+
                reverse_index_bits_kernel_use+
                hash_leaves_kernel_use+
                reduce_digests_kernel_use+
                transpose_kernel_use;

        printf("total use:%.2lf\n", total_use);
        return RustError{hipSuccess};
    }

    RustError merkle_tree_from_coeffs(
            GoldilocksField* d_values_flatten,
            GoldilocksField* d_ext_values_flatten,
            int poly_num, int values_num_per_poly, int log_len,
            const GoldilocksField* d_root_table, const GoldilocksField* d_root_table2, const GoldilocksField* d_shift_powers,
            int rate_bits, int salt_size,
            int cap_height,
            int pad_extvalues_len,
            CudaInvContext* ctx
    ){
        printf("start merkle_tree_from_coeffs: poly_num:%d, values_num_per_poly:%d, log_len:%d\n",
               poly_num, values_num_per_poly, log_len);
        printf("d_values_flatten: %p, d_ext_values_flatten: %p\n", d_values_flatten, d_ext_values_flatten);

        int values_num_per_extpoly = values_num_per_poly*(1<<rate_bits);
        auto stream = ctx->stream;
        int ext_poly_num = poly_num + salt_size;

        int len_cap = 1 << cap_height;
        int num_digests = 2 * (values_num_per_extpoly - len_cap);

        int thcnt = 0;
        int nthreads = 32;
        double lde_kernel_use, mul_shift_kernel_use, fft_kernel_use, reverse_index_bits_kernel_use,
                hash_leaves_kernel_use, reduce_digests_kernel_use, transpose_kernel_use;

        d_ext_values_flatten += pad_extvalues_len;


        clock_t start;

        start = clock();
        thcnt = values_num_per_poly*poly_num;
        nthreads = 32;
        lde_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_values_flatten, d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits);
        hipStreamSynchronize(stream);
        printf("lde_kernel elapsed: %.2lf\n", lde_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly*poly_num;
        nthreads = 32;
        init_lde_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits);
        hipStreamSynchronize(stream);
        printf("init_lde_kernel elapsed: %.2lf\n", (double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_poly*poly_num;
        nthreads = 32;
        mul_shift_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits, d_shift_powers);
        hipStreamSynchronize(stream);
        printf("mul_shift_kernel elapsed: %.2lf\n", mul_shift_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        fft_kernel<<<poly_num, 32*8, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly*(1<<rate_bits), log_len+rate_bits, d_root_table2, rate_bits);
        hipStreamSynchronize(stream);
        printf("fft_kernel elapsed: %.2lf\n", fft_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        thcnt = values_num_per_extpoly*poly_num;
        nthreads = 32;
        reverse_index_bits_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_extpoly, log_len+rate_bits);
        hipStreamSynchronize(stream);
        printf("reverse_index_bits_kernel elapsed: %.2lf\n", reverse_index_bits_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        int log2_leaves_len = log_len + rate_bits;
        assert(cap_height <= log2_leaves_len);

        auto *d_digest_buf = (PoseidonHasher::HashOut*)(d_ext_values_flatten + values_num_per_extpoly * ext_poly_num);

        start = clock();
        thcnt = values_num_per_extpoly;
        nthreads = 32;
        hash_leaves_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(
                d_ext_values_flatten, poly_num+salt_size, values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
        hipStreamSynchronize(stream);
        printf("hash_leaves_kernel elapsed: %.2lf\n", hash_leaves_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        start = clock();
        nthreads = 32*8;
        thcnt = len_cap * nthreads;
        reduce_digests_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
        hipStreamSynchronize(stream);
        printf("reduce_digests_kernel elapsed: %.2lf\n", reduce_digests_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

        hipStreamSynchronize(ctx->stream2);

        start = clock();
        thcnt = values_num_per_extpoly;
        nthreads = 32;
        transpose_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, d_ext_values_flatten - pad_extvalues_len, ext_poly_num, values_num_per_extpoly);
        hipStreamSynchronize(stream);
        printf("transpose_kernel elapsed: %.2lf\n", transpose_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);
        
       /* GoldilocksField * leaves = d_ext_values_flatten -pad_extvalues_len; */
       /* for (int i =0; i < ext_poly_num; ++i) { */
       /*      printf("leaves: "); */
       /*      for (int j=0; j < values_num_per_extpoly; ++j) { */
       /*      printf("%lld, ", leaves+(i * values_num_per_extpoly + j)); */
       /*      } */
       /*      printf("\n"); */
       /* } */

        double total_use =
                lde_kernel_use+
                mul_shift_kernel_use+
                fft_kernel_use+
                reverse_index_bits_kernel_use+
                hash_leaves_kernel_use+
                reduce_digests_kernel_use+
                transpose_kernel_use;

        printf("total use:%.2lf\n", total_use);
        return RustError{hipSuccess};
    }

}

#endif
